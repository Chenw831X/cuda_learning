#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <cstdio>

#include "test.cuh"


__global__ void addMatrix(double *a, int row, int col){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= row){
        return;
    }
    for(int i=0; i<col; ++i){
        a[idx+i*row] += 0.5;
    }
}

void func(Eigen::MatrixXd &a){
    int dev = 0;
    hipSetDevice(dev);

    int row = a.rows(), col = a.cols();
    int size = a.size();
    double *a_h = a.data();
    double *a_d = NULL;
    CHECK(hipMalloc((double**)&a_d, size*sizeof(double)));
    CHECK(hipMemcpy(a_d, a_h, size*sizeof(double), hipMemcpyHostToDevice));

    dim3 block(1024);
    dim3 grid((row-1)/block.x+1);
    addMatrix<<<grid, block>>>(a_d, row, col);
    hipDeviceSynchronize();

    CHECK(hipMemcpy(a_h, a_d, size*sizeof(double), hipMemcpyDeviceToHost));
    hipFree(a_d);
    hipDeviceReset();
}
