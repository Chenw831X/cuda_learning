#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <cstdio>

#include "addArray.cuh"

__global__ void addArray(int *a, int *b, int *c, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= n){
        return;
    }
    c[idx] = a[idx] + b[idx];
}

void func(int *a, int *b, int *c, int n){
    int dev = 0;
    hipSetDevice(dev);

    int *a_d = NULL;
    int *b_d = NULL;
    int *c_d = NULL;
    CHECK(hipMalloc((int**)&a_d, n*sizeof(int)));
    CHECK(hipMalloc((int**)&b_d, n*sizeof(int)));
    CHECK(hipMalloc((int**)&c_d, n*sizeof(int)));

    CHECK(hipMemcpy(a_d, a, n*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d, b, n*sizeof(int), hipMemcpyHostToDevice));

    dim3 block(1024);
    dim3 grid((n-1)/block.x+1);
    addArray<<<grid, block>>>(a_d, b_d, c_d, n);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(c, c_d, n*sizeof(int), hipMemcpyDeviceToHost));

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipDeviceReset();
}