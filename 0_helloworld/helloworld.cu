#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime_api.h"

__global__ void hello_world(void){
    printf("GPU: hello world\n");
}

int main(int argc, char **argv){
    printf("CPU: hello world\n");
    double tBegin = cpuSecond();
    hello_world<<<1, 10>>>();
    hipDeviceReset();
    double tElapsed = cpuSecond() - tBegin;
    printf("Time elapsed: %f\n", tElapsed);
    return 0;
}